#include <stdio.h>
#include <hip/hip_runtime.h>


#define DEBUG 0

//will compute local histogram
//assuming passed pointers are adjusted for the thread
//bitpos is the lsb from which to consider numbits towards msb
__device__ void computeLocalHisto(int *localHisto, float *arrElem, int n,
				  int numBits, int bitpos) {
  
  int i;
  int numBuckets = 1 << numBits;
  int mask = (1 << numBits) - 1;
  int key;

  for (i = 0; i < n; i++) {
    key = (((int)arrElem[i]) >> bitpos) & mask;
    localHisto[key]++;
  }
  
}



__device__ void dispArr(int *arr, int n) {
  
  int i;

  //threadId with in a block, DMat doc to start with
  int thId = threadIdx.x; 
  

  if (thId == 0) {
    printf("\n");
    for (i = 0; i < n; i++) {
      printf(" %d ", arr[i]);
    }
    printf("\n");
  }
}


//assuming sizeof int == sizeof float
__device__ void computeAtomicHisto(int *aggHisto, float *arrElem, int numElem,
				   int numBits, int bitpos) {
  
  int i, j;
  int numBuckets = 1 << numBits;
  int mask = (1 << numBits) - 1;
  int key;
  void *vptr;
  int *iptr;
  //thread id within a block
  int threadId = threadIdx.x;
  
  //number of threads in block
  int nThreads = blockDim.x;

  for (i = threadId; i < numElem; i+=nThreads) {
    vptr = (void*)(arrElem + i);
    iptr = (int*)vptr;
    key =   ( (*iptr) >> bitpos)  & mask;
    atomicAdd(&(aggHisto[key]), 1);
  }

}

//assuming sizeof int == sizeof float
__device__ void writeSortedVals(int *aggHisto, float *fromArr, float *toArr,
				int numBits, int bitpos, int n) {
  int i, key;
  int mask = (1 << numBits) - 1;
  void *vptr;
  int *iptr;

  for (i = 0; i < n; i++) {
    vptr = (void*)(fromArr + i);
    iptr = (int*)vptr;

    key = (  (*iptr) >> bitpos) & mask;

    if (DEBUG) {
      printf("toArr[%d] = %f\n", aggHisto[key], fromArr[i]);
    }

    toArr[aggHisto[key]++] = fromArr[i];
  }
}


__device__ void zeroedInt(int *arr, int count) {
  int i;
  
  //thread id within a block
  int threadId = threadIdx.x;
  
  //number of threads in block
  int nThreads = blockDim.x;

  for (i = threadId; i < count; i+=nThreads) {
    arr[i] = 0;
  }
}



//scan array arr of size n=nThreads, power of 2
__device__ void preSubScan(int *arr, int n, int prev) {

  int i, d, ai, bi, offset, temp;
  //threadId with in a block, DMat doc to start with
  int thId = threadIdx.x; 

  //number of threads in blocks
  int nThreads = blockDim.x;

  d = 0;
  offset = 1;

  //build sum in place up the tree
  for (d = n>>1; d > 0; d >>=1) {
    __syncthreads();
    if (thId < d) {
      ai = offset*(2*thId+1) - 1;
      bi = offset*(2*thId+2) - 1;
      arr[bi] += arr[ai];
    }
    offset*=2;
  }
  
  //clear last element
  if (thId == 0) {
    arr[n-1] = 0;
  }

  //traverse down tree & build scan
  for (int d = 1; d < n; d *=2) {
    offset = offset >> 1;
    __syncthreads();
    if (thId < d) {
      ai = offset*(2*thId + 1) - 1;
      bi = offset*(2*thId + 2) - 1;
      temp = arr[ai];
      arr[ai] = arr[bi];
      arr[bi] += temp;
    }
  }

  for (i = thId; i < n; i+=nThreads) {
    arr[i] += prev;
  }

  __syncthreads();
}



__device__ void d_dispFArr(float *arr, int n) {
  int i;

  //threadId with in a block, DMat doc to start with
  int thId = threadIdx.x; 
  
  if (thId == 0) {
    printf("\n");
    for (i = 0; i < n; i++) {
      printf(" %f ", arr[i]);
    }
    printf("\n");
  }

}


//works efficiently for power of 2
__device__ void scan(int *arr, int n) {
  
  int i, j, prev, next, temp;

  //threadId with in a block, DMat doc to start with
  int thId = threadIdx.x; 

  //number of threads in blocks
  int nThreads = blockDim.x;


  //divide the simpred into nThreads blocks,
  //scan each block in parallel, with next iteration using results from prev blocks
  prev = 0;
  next = 0;

  for (i = 0; i < n; i += nThreads) {
    //dispArr(arr, n);
    next = arr[i+nThreads-1];
    if (n - i >= nThreads) {
      preSubScan(arr + i, nThreads, (i>0?arr[i-1]:0) + prev);
    } else {
      //not power of 2 perform serial scan for others
      //this will be last iteration of loop
      if (thId == 0) {
	for (j = i; j < n; j++) {
	  temp = prev + arr[j-1];
	  prev = arr[j];
	  arr[j] = temp;
	}
      }      
    }//end else
    
    prev = next;

  }//end for

  __syncthreads();
} 




//numbits means bits at a time
__global__ void radixSort(float *d_InArr, int n, int numBits) {
  
  int i, j, elemPerThread;
  int localHistoElemCount;

  //get current block number
  int blockId = blockIdx.x;
  
  //thread id within a block
  int threadId = threadIdx.x;

  //number of threads in block
  int nThreads = blockDim.x;

  //global thread id
  int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

  extern __shared__ int s[];

  //shared mem space for aggregated histogram
  int *aggHisto = s;

  //shared mem space to copy array to be sorted
  float *fromArr = (float*) &aggHisto[1<<numBits];
  float *toArr = (float *) &fromArr[n];
  float *tempSwap;

  //bucket size
  int bucketSize = 1 << numBits;

  //initialize arrays in shared mem
  for (i = threadId; i < n; i+=nThreads) {
    fromArr[i] = d_InArr[i];
    toArr[i] = 0;
  }
  
  if (threadId == 0 && DEBUG) {
    printf("\n fromArray:  ");
    d_dispFArr(fromArr, n);
  }


  //for each numbits chunk do following
  for (i = 0; i < sizeof(float)*8; i+=numBits) {
    //reset histogram
    zeroedInt(aggHisto, bucketSize);

    if (threadId == 0 && DEBUG) {
      printf("\n fromArray b4 histo :  ");
      d_dispFArr(fromArr, n);
    }

    //aggregate in histogram in shared mem
    computeAtomicHisto(aggHisto, fromArr, n,
		       numBits, i);

    if (threadId == 0 && DEBUG) {
      printf("\naggHisto, bitpos:%d:", i);
      dispArr(aggHisto, bucketSize);
      printf("\n fromArray after histo :  ");
      d_dispFArr(fromArr, n);
    }
    
    //perform scan on aggHisto (assuming power of 2)
    scan(aggHisto, bucketSize);

    if (threadId == 0 && DEBUG) {
      printf("\naggHisto after scan, bitpos:%d:", i);
      dispArr(aggHisto, bucketSize);
    }

    __syncthreads();

    if (threadId == 0) {
      //copy values to correct output by a single thread
      writeSortedVals(aggHisto, fromArr, toArr,
		      numBits, i, n);

    }
    __syncthreads();

    if (threadId == 0 && DEBUG) {
      printf("\n sorted:  ");
      d_dispFArr(toArr, n);
    }

    //toArr contains the sorted arr, for the next iteration point fromArr to this location
    tempSwap = toArr;
    toArr = fromArr;
    fromArr = tempSwap;  
  }

  //at this point fromAr will contain sorted arr in mem
  //write this out to device in parallel
  for (i = threadId; i < n; i+=nThreads) {
    d_InArr[i] = fromArr[i];
  }
  

}


void dispFArr(float *arr, int n) {
  int i;
  for (i = 0; i < n; i++) {
    printf(" %f ", arr[i]);
  }
}


int main(int argc, char *argv[]) {

  float h_fArr[] = {0.1, 0, 0.5, 0.8, 0, 0.7, 0.8, 1.3, 0.0, 2.5, 9.10, 0, 2};
  int h_n = 13;

  //float h_fArr[] = {0.1, 0.6, 0.4, 0.3, 0.8, 2.0};
  //int h_n = 6;


  float *d_fArr;
  float *h_fSortedArr;

  int i;
  int numBits = 2;

  printf("\n");
  dispFArr(h_fArr, h_n);

  
  //allocate mem on device
  hipMalloc((void **) &d_fArr, sizeof(float)*h_n);
  
  //copy to device
  hipMemcpy((void *) d_fArr, (void *) h_fArr, sizeof(float)*h_n, hipMemcpyHostToDevice);

  //sort with 2 bits at a time
  radixSort<<<1, 4, (sizeof(int)*(1<<numBits) + sizeof(float)*h_n*2)>>>(d_fArr, h_n, numBits);
  
  //copy sorted back to host
  hipMemcpy((void *)h_fArr , (void *) d_fArr, sizeof(float)*h_n, hipMemcpyDeviceToHost);
  
  printf("\n");
  dispFArr(h_fArr, h_n);
  printf("\n");
}
